#include "hip/hip_runtime.h"
////
////
//// Compile with: nvcc -o main main.cu -lnppif `pkg-config opencv --cflags --libs`
////
////


#include <hip/hip_runtime.h>
#include <npp.h>

#include "opencv2/core.hpp"
#include "opencv2/imgproc.hpp"
#include "opencv2/highgui.hpp"

int main( int argc, char *argv[] )
{

    ///
    /// Set test filename
    ///
    const std::string inputFile = "test.png";

    //----------------------------------------------------------------------

    ///
    /// Read the input image on host
    ///
    cv::Mat inputImage = cv::imread( inputFile, CV_LOAD_IMAGE_GRAYSCALE ); // NOTE type of image to read

    assert( inputImage.data && "Could not open or find input image file" );

    const int mW = inputImage.cols;
    const int mH = inputImage.rows;
    printf( "The size of the input image: (%d,%d)\n", mW, mH );

    //----------------------------------------------------------------------

    ///
    /// Allocate memory on device
    ///
    Npp8u  *d_iImage;
    Npp32u *d_tImage;
    hipMalloc( (void **)(&d_iImage), mW*mH*sizeof(Npp8u)  );
    hipMalloc( (void **)(&d_tImage), mW*mH*sizeof(Npp32u) );

    //----------------------------------------------------------------------

    ///
    /// Transfer data to device
    ///
    hipMemcpy( d_iImage, inputImage.data, mW*mH*sizeof(Npp8u), hipMemcpyHostToDevice );

    //----------------------------------------------------------------------

    int max;
    NppiSize srcSize = { mW, mH };
    NppStatus npp_err;

    ///
    /// Get buffer size
    ///
    int nBufferSize = 0;
    npp_err = nppiLabelMarkersGetBufferSize_8u32u_C1R( srcSize, &nBufferSize );
    assert( npp_err == NPP_SUCCESS );

    // Allocate the scratch buffer 
    Npp8u *pBuffer = 0;
    hipMalloc( (void **)(&pBuffer), nBufferSize );

    printf( "Buffer size 1: %d\n", nBufferSize );

    ///
    /// Connected components labelling
    ///
    npp_err = nppiLabelMarkers_8u32u_C1R(
            d_iImage,
            mW*sizeof(Npp8u),
            d_tImage,
            mW*sizeof(Npp32u),
            srcSize,
            (Npp8u)0,
            nppiNormInf, // 8-way connectivity
            &max,
            pBuffer
            );
    assert( npp_err == NPP_SUCCESS );

    printf( "Max 1: %d\n", max );

    //-----------------------------------------------------------------------------------------

    ///
    /// Get buffer size
    ///
    int nCompressBufferSize = 0;
    npp_err = nppiCompressMarkerLabelsGetBufferSize_32u8u_C1R( max, &nCompressBufferSize );
    assert( npp_err == NPP_SUCCESS );
    
    if( nCompressBufferSize > nBufferSize ) {
        nBufferSize = nCompressBufferSize;
        hipFree( pBuffer );
        hipMalloc( &pBuffer, nBufferSize );
    }

    printf( "Buffer size 2: %d\n", nBufferSize );

    ///
    /// Compress marker labels
    ///
    npp_err = nppiCompressMarkerLabels_32u8u_C1R(
            d_tImage,
            mW*sizeof(Npp32u),
            d_iImage,
            mW*sizeof(Npp8u),
            srcSize,
            max,
            &max,
            pBuffer
            );
    
    assert( npp_err == NPP_SUCCESS );

    printf( "Max 2: %d\n", max );

    assert( max < 256 && "Number of connected components found exceeds limit." );
    
    // Transfer output to host
    cv::Mat h_oImage( mH, mW, 0 );
    hipMemcpy( h_oImage.data, d_iImage, mW*mH*sizeof(Npp8u), hipMemcpyDeviceToHost );

    cv::imwrite( "output.png", h_oImage );

    //----------------------------------------------------------------------

    ///
    /// Clean up
    ///
    hipFree( d_iImage );
    hipFree( d_tImage );
    hipFree( pBuffer );

    return 0;

}
